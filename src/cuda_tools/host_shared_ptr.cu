#include "hip/hip_runtime.h"
#include "host_shared_ptr.cuh"
#include "cuda_tools/device_buffer.cuh"

#include <hip/hip_runtime.h>
#include <cstdio>

#include "cuda_error_checking.cuh"
#include "template_generator.hh"

namespace cuda_tools
{

template_generation(host_shared_ptr);

template <typename T>
__host__
void host_shared_ptr<T>::allocate(std::size_t size)
{
    cuda_safe_call(hipMalloc((void**)&data_, sizeof(T) * size));
}

template <typename T>
host_shared_ptr<T>::host_shared_ptr(std::size_t size) : size_(size)
{
    allocate(size);
}

template <typename T>
host_shared_ptr<T>::host_shared_ptr(host_shared_ptr<T>&& ptr) : data_(ptr.data_), size_(ptr.size_), counter_(ptr.counter_ + 1)
{}

template <typename T>
host_shared_ptr<T>::host_shared_ptr(host_shared_ptr<T>& ptr) : data_(ptr.data_), size_(ptr.size_), counter_(ptr.counter_ + 1)
{}

template <typename T>
host_shared_ptr<T>& host_shared_ptr<T>::operator=(host_shared_ptr<T>&& r)
{
    data_ = r.data_;
    size_ = r.size_;
    counter_ = r.counter_ + 1;
    return *this;
}

template <typename T>
host_shared_ptr<T>::~host_shared_ptr()
{
    if (--counter_ == 0)
    {
        cuda_safe_call(hipFree(data_));
        if (host_data_ != nullptr)
            delete[] host_data_;
    }
}

template <typename T>
T* host_shared_ptr<T>::download()
{
    if (data_ != nullptr)
    {
        if (host_data_ == nullptr)
            host_data_ = new T[size_];
        cuda_safe_call(hipMemcpy(host_data_, data_, sizeof(T) * size_, hipMemcpyDeviceToHost));
    }
    return host_data_;
}

template <typename T, typename FUNC>
__global__
static void kernel_fill(cuda_tools::device_buffer<T> buffer, FUNC func)
{
    const int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < buffer.size_)
        buffer[index] = func();
}

template <typename T>
void host_shared_ptr<T>::fill(const T val)
{
    constexpr int TILE_WIDTH  = 64;
    constexpr int TILE_HEIGHT = 1;

    cuda_tools::device_buffer<T> device_buffer(*this);

    auto lambda = [val] __device__ { return val; };

    const int gx = (this->size_ + TILE_WIDTH - 1) / TILE_WIDTH;
    const int gy = 1;

    const dim3 block(TILE_WIDTH, TILE_HEIGHT);
    const dim3 grid(gx, gy);

    kernel_fill<T><<<grid, block>>>(device_buffer, lambda);
    kernel_check_error();

    hipDeviceSynchronize();
}


} // namespace cuda_tools